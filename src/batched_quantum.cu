#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Regis Araujo Melo
 * This program is free software under the GPL-3.0 license. See LICENSE file.
 */

#include "batched_quantum.cuh"
#include <hip/hip_runtime.h>
#include <fmt/core.h>
#include <fmt/color.h>

namespace ohmy {
namespace quantum {

// ============================================================================
// BATCHED KERNELS - Process multiple state vectors in parallel
// ============================================================================

/**
 * @brief Initialize all states in batch to |0⟩^⊗n
 * 
 * Grid: 2D with blockIdx.y = batch index
 */
__global__ void initialize_states_batched(
    Complex* states,      // [batch_size][state_size]
    int batch_size,
    size_t state_size
) {
    int batch_idx = blockIdx.y;
    if (batch_idx >= batch_size) return;
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= state_size) return;
    
    // Offset to this batch's state vector
    Complex* state = states + batch_idx * state_size;
    
    if (idx == 0) {
        state[0] = make_hipDoubleComplex(1.0, 0.0);  // |0⟩
    } else {
        state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
}

/**
 * @brief Batched RY gate
 */
__global__ void apply_ry_gate_batched(
    Complex* states,
    int batch_size,
    int target_qubit,
    const double* angles,  // [batch_size] - one angle per batch
    int num_qubits
) {
    int batch_idx = blockIdx.y;
    if (batch_idx >= batch_size) return;
    
    size_t state_size = 1ULL << num_qubits;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= state_size) return;
    
    Complex* state = states + batch_idx * state_size;
    __shared__ double s_cos_half;
    __shared__ double s_sin_half;
    if (threadIdx.x == 0) {
        double angle = angles[batch_idx];
        s_cos_half = cos(angle / 2.0);
        s_sin_half = sin(angle / 2.0);
    }
    __syncthreads();
    
    // Standard RY gate logic (same as non-batched)
    if ((idx & (1ULL << target_qubit)) != 0) return;
    
    size_t flipped_idx = idx | (1ULL << target_qubit);
    
    Complex a0 = state[idx];
    Complex a1 = state[flipped_idx];
    
    state[idx] = hipCadd(
        make_hipDoubleComplex(s_cos_half * hipCreal(a0), s_cos_half * hipCimag(a0)),
        make_hipDoubleComplex(-s_sin_half * hipCreal(a1), -s_sin_half * hipCimag(a1))
    );
    
    state[flipped_idx] = hipCadd(
        make_hipDoubleComplex(s_sin_half * hipCreal(a0), s_sin_half * hipCimag(a0)),
        make_hipDoubleComplex(s_cos_half * hipCreal(a1), s_cos_half * hipCimag(a1))
    );
}

/**
 * @brief Batched RZ gate
 */
__global__ void apply_rz_gate_batched(
    Complex* states,
    int batch_size,
    int target_qubit,
    const double* angles,  // [batch_size]
    int num_qubits
) {
    int batch_idx = blockIdx.y;
    if (batch_idx >= batch_size) return;
    
    size_t state_size = 1ULL << num_qubits;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= state_size) return;
    
    Complex* state = states + batch_idx * state_size;
    __shared__ double s_cos_half;
    __shared__ double s_sin_half;
    if (threadIdx.x == 0) {
        double angle = angles[batch_idx];
        s_cos_half = cos(angle / 2.0);
        s_sin_half = sin(angle / 2.0);
    }
    __syncthreads();

    // phase = +/- angle/2 -> cos(phase)=cos(angle/2); sin(phase)=sign * sin(angle/2)
    int sign = ((idx & (1ULL << target_qubit)) != 0) ? 1 : -1;
    Complex rotation = make_hipDoubleComplex(s_cos_half, sign * s_sin_half);
    state[idx] = hipCmul(state[idx], rotation);
}

/**
 * @brief Batched CNOT gate
 */
__global__ void apply_cnot_gate_batched(
    Complex* states,
    int batch_size,
    int control_qubit,
    int target_qubit,
    int num_qubits
) {
    int batch_idx = blockIdx.y;
    if (batch_idx >= batch_size) return;
    
    size_t state_size = 1ULL << num_qubits;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= state_size) return;
    
    Complex* state = states + batch_idx * state_size;
    
    // CNOT: flip target if control is |1⟩
    if ((idx & (1ULL << control_qubit)) == 0) return;
    if ((idx & (1ULL << target_qubit)) != 0) return;
    
    size_t flipped_idx = idx | (1ULL << target_qubit);
    
    if (idx < flipped_idx) {
        Complex temp = state[idx];
        state[idx] = state[flipped_idx];
        state[flipped_idx] = temp;
    }
}

/**
 * @brief Batched fused RY+RZ layer (OPTIMIZED)
 * 
 * Applies RY followed by RZ to all qubits in one kernel.
 * Reduces 32 kernel launches to 16 per batch.
 */
__global__ void apply_fused_ry_rz_layer_batched(
    Complex* states,
    int batch_size,
    const double* ry_angles,  // [batch_size][16]
    const double* rz_angles,  // [batch_size][16]
    int num_qubits
) {
    int batch_idx = blockIdx.y;
    if (batch_idx >= batch_size) return;
    
    size_t state_size = 1ULL << num_qubits;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= state_size) return;
    
    Complex* state = states + batch_idx * state_size;
    const double* ry = ry_angles + batch_idx * num_qubits;
    const double* rz = rz_angles + batch_idx * num_qubits;
    
    // Apply all 16 RY gates sequentially, then all 16 RZ gates
    // This maintains determinism while processing entire layer
    
    // RY layer
    for (int q = 0; q < num_qubits; q++) {
        if ((idx & (1ULL << q)) != 0) continue;
        
        size_t flipped_idx = idx | (1ULL << q);
        double angle = ry[q];
        double cos_half = cos(angle / 2.0);
        double sin_half = sin(angle / 2.0);
        
        Complex a0 = state[idx];
        Complex a1 = state[flipped_idx];
        
        Complex new_a0 = hipCadd(
            make_hipDoubleComplex(cos_half * hipCreal(a0), cos_half * hipCimag(a0)),
            make_hipDoubleComplex(-sin_half * hipCreal(a1), -sin_half * hipCimag(a1))
        );
        
        Complex new_a1 = hipCadd(
            make_hipDoubleComplex(sin_half * hipCreal(a0), sin_half * hipCimag(a0)),
            make_hipDoubleComplex(cos_half * hipCreal(a1), cos_half * hipCimag(a1))
        );
        
        state[idx] = new_a0;
        state[flipped_idx] = new_a1;
    }
    
    // RZ layer
    for (int q = 0; q < num_qubits; q++) {
        double angle = rz[q];
        double phase = ((idx & (1ULL << q)) != 0) ? angle / 2.0 : -angle / 2.0;
        Complex rotation = make_hipDoubleComplex(cos(phase), sin(phase));
        state[idx] = hipCmul(state[idx], rotation);
    }
}

/**
 * @brief Batched measurement kernel
 * 
 * Computes ⟨σz⟩ for all qubits in all states
 */
__global__ void measure_expectations_batched(
    const Complex* states,
    int batch_size,
    double* expectations,  // [batch_size][num_qubits]
    int num_qubits
) {
    int batch_idx = blockIdx.z;  // Use 3D grid for measurement
    int qubit = blockIdx.y;
    
    if (batch_idx >= batch_size || qubit >= num_qubits) return;
    
    size_t state_size = 1ULL << num_qubits;
    const Complex* state = states + batch_idx * state_size;
    
    __shared__ double block_sum;
    if (threadIdx.x == 0) block_sum = 0.0;
    __syncthreads();
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    double thread_sum = 0.0;
    if (idx < state_size) {
        Complex amplitude = state[idx];
        double prob = hipCreal(amplitude) * hipCreal(amplitude) +
                     hipCimag(amplitude) * hipCimag(amplitude);
        
        // Expectation: +1 if qubit is |0⟩, -1 if qubit is |1⟩
        int sign = ((idx & (1ULL << qubit)) == 0) ? 1 : -1;
        thread_sum = sign * prob;
    }
    
    // Atomic add to shared memory
    atomicAdd(&block_sum, thread_sum);
    __syncthreads();
    
    // Write result
    if (threadIdx.x == 0) {
        atomicAdd(&expectations[batch_idx * num_qubits + qubit], block_sum);
    }
}

// ============================================================================
// BatchedQuantumSimulator Implementation
// ============================================================================

namespace {
// Toggle for experimental fused RY+RZ layer kernel. Disabled by default
// because it may reduce performance on some GPUs and complicates memory
// access patterns. Enable only after benchmarking.
constexpr bool kEnableLayerFusion = false;
}

BatchedQuantumSimulator::BatchedQuantumSimulator(int num_qubits, int batch_size)
    : num_qubits_(num_qubits)
    , batch_size_(batch_size)
    , state_size_(1ULL << num_qubits)
    , d_states_(nullptr)
    , d_expectations_(nullptr)
{
    // Allocate GPU memory for all state vectors
    size_t total_state_size = batch_size_ * state_size_ * sizeof(Complex);
    size_t total_expectations_size = batch_size_ * num_qubits_ * sizeof(double);
    
    hipError_t err = hipMalloc(&d_states_, total_state_size);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Failed to allocate {} MB for batched states: {}\n",
            total_state_size / (1024 * 1024), hipGetErrorString(err));
        throw std::runtime_error("GPU memory allocation failed");
    }
    
    err = hipMalloc(&d_expectations_, total_expectations_size);
    if (err != hipSuccess) {
        hipFree(d_states_);
        fmt::print(fg(fmt::color::red),
            "Failed to allocate expectations memory: {}\n", hipGetErrorString(err));
        throw std::runtime_error("GPU memory allocation failed");
    }
    
    // Allocate reusable angles buffer (per-gate)
    err = hipMalloc(&d_angles_, batch_size_ * sizeof(double));
    if (err != hipSuccess) {
        hipFree(d_states_);
        hipFree(d_expectations_);
        fmt::print(fg(fmt::color::red),
            "Failed to allocate angles buffer: {}\n", hipGetErrorString(err));
        throw std::runtime_error("GPU memory allocation failed");
    }

    // Allocate persistent fused-layer angle buffers [batch_size_ * num_qubits_]
    err = hipMalloc(&d_layer_ry_, batch_size_ * num_qubits_ * sizeof(double));
    if (err != hipSuccess) {
        hipFree(d_states_);
        hipFree(d_expectations_);
        hipFree(d_angles_);
        fmt::print(fg(fmt::color::red),
            "Failed to allocate fused RY buffer: {}\n", hipGetErrorString(err));
        throw std::runtime_error("GPU memory allocation failed");
    }

    err = hipMalloc(&d_layer_rz_, batch_size_ * num_qubits_ * sizeof(double));
    if (err != hipSuccess) {
        hipFree(d_states_);
        hipFree(d_expectations_);
        hipFree(d_angles_);
        hipFree(d_layer_ry_);
        fmt::print(fg(fmt::color::red),
            "Failed to allocate fused RZ buffer: {}\n", hipGetErrorString(err));
        throw std::runtime_error("GPU memory allocation failed");
    }

    // Allocate host pinned buffers
    err = hipHostAlloc(&h_gate_angles_, batch_size_ * sizeof(double), hipHostMallocDefault);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Failed to allocate pinned host gate angles: {}\n", hipGetErrorString(err));
        throw std::runtime_error("Pinned host allocation failed");
    }

    err = hipHostAlloc(&h_layer_ry_, batch_size_ * num_qubits_ * sizeof(double), hipHostMallocDefault);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Failed to allocate pinned host layer RY: {}\n", hipGetErrorString(err));
        throw std::runtime_error("Pinned host allocation failed");
    }

    err = hipHostAlloc(&h_layer_rz_, batch_size_ * num_qubits_ * sizeof(double), hipHostMallocDefault);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Failed to allocate pinned host layer RZ: {}\n", hipGetErrorString(err));
        throw std::runtime_error("Pinned host allocation failed");
    }

    err = hipHostAlloc(&h_expectations_pinned_, batch_size_ * num_qubits_ * sizeof(double), hipHostMallocDefault);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Failed to allocate pinned host expectations: {}\n", hipGetErrorString(err));
        throw std::runtime_error("Pinned host allocation failed");
    }

    fmt::print("Batched quantum simulator initialized: {} qubits, batch size = {}\n",
        num_qubits_, batch_size_);
    fmt::print("  GPU memory: {} MB states + {} KB expectations = {} MB total\n",
        total_state_size / (1024 * 1024),
        total_expectations_size / 1024,
        (total_state_size + total_expectations_size) / (1024 * 1024));
}

BatchedQuantumSimulator::~BatchedQuantumSimulator() {
    if (d_states_) hipFree(d_states_);
    if (d_expectations_) hipFree(d_expectations_);
    if (d_angles_) hipFree(d_angles_);
    if (d_layer_ry_) hipFree(d_layer_ry_);
    if (d_layer_rz_) hipFree(d_layer_rz_);
    if (h_gate_angles_) hipHostFree(h_gate_angles_);
    if (h_layer_ry_) hipHostFree(h_layer_ry_);
    if (h_layer_rz_) hipHostFree(h_layer_rz_);
    if (h_expectations_pinned_) hipHostFree(h_expectations_pinned_);
}

bool BatchedQuantumSimulator::initialize_states() {
    int block_size = 512;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    
    dim3 grid(num_blocks, batch_size_);
    dim3 block(block_size);
    
    initialize_states_batched<<<grid, block, 0, stream_>>>(d_states_, batch_size_, state_size_);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Batched initialization error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    return true;
}

bool BatchedQuantumSimulator::apply_circuits_optimized(
    const std::vector<QuantumCircuit>& circuits)
{
    if (circuits.size() != static_cast<size_t>(batch_size_)) {
        fmt::print(fg(fmt::color::red),
            "Circuit count mismatch: expected {}, got {}\n",
            batch_size_, circuits.size());
        return false;
    }
    
    // For now, use simple approach: apply gates one at a time across batch
    // Future optimization: group gates and apply in parallel
    
    int block_size = 512;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    dim3 grid(num_blocks, batch_size_);
    dim3 block(block_size);
    
    // Assume all circuits have same structure (QTC property)
    const auto& reference_circuit = circuits[0];
    
    size_t gate_idx = 0;
    const size_t total_gates = reference_circuit.gates.size();
    
    while (gate_idx < total_gates) {
        // Try to fuse a layer: [RY over all qubits][RZ over all qubits]
        bool can_fuse_layer = kEnableLayerFusion ? false : false; // default disabled
        if (kEnableLayerFusion) {
            can_fuse_layer = false;
        }
        if (gate_idx + (2 * num_qubits_) - 1 < total_gates) {
            can_fuse_layer = true;
            // Check first num_qubits_ are RY on distinct targets 0..num_qubits_-1
            for (int q = 0; q < num_qubits_; ++q) {
                const auto& g = reference_circuit.gates[gate_idx + q];
                if (g.type != GateType::RY || g.target_qubit != q) { can_fuse_layer = false; break; }
            }
            // Check next num_qubits_ are RZ on same targets
        if (can_fuse_layer) {
                for (int q = 0; q < num_qubits_; ++q) {
                    const auto& g = reference_circuit.gates[gate_idx + num_qubits_ + q];
                    if (g.type != GateType::RZ || g.target_qubit != q) { can_fuse_layer = false; break; }
                }
            }
        }

    if (can_fuse_layer) {
            // Build [batch][num_qubits] ry and rz angle matrices in pinned host buffers
            for (int b = 0; b < batch_size_; ++b) {
                for (int q = 0; q < num_qubits_; ++q) {
                    h_layer_ry_[b * num_qubits_ + q] = circuits[b].gates[gate_idx + q].angle;
                    h_layer_rz_[b * num_qubits_ + q] = circuits[b].gates[gate_idx + num_qubits_ + q].angle;
                }
            }

            // Copy layer angles into persistent device buffers (async, pinned host)
            hipMemcpyAsync(d_layer_ry_, h_layer_ry_, batch_size_ * num_qubits_ * sizeof(double), hipMemcpyHostToDevice, stream_);
            hipMemcpyAsync(d_layer_rz_, h_layer_rz_, batch_size_ * num_qubits_ * sizeof(double), hipMemcpyHostToDevice, stream_);

            apply_fused_ry_rz_layer_batched<<<grid, block, 0, stream_>>>(
                d_states_, batch_size_, d_layer_ry_, d_layer_rz_, num_qubits_);

            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                fmt::print(fg(fmt::color::red),
                    "Batched fused layer error at gates [{}..{}]: {}\n",
                    gate_idx, gate_idx + 2 * num_qubits_ - 1, hipGetErrorString(err));
                return false;
            }

            gate_idx += 2 * num_qubits_;
            continue;
        }

        // No fusion possible: apply single gate across batch
        const auto& gate = reference_circuit.gates[gate_idx];
        if (gate.type == GateType::CNOT) {
            apply_cnot_gate_batched<<<grid, block, 0, stream_>>>(
                d_states_, batch_size_, gate.control_qubit,
                gate.target_qubit, num_qubits_);
        } else {
            for (int b = 0; b < batch_size_; b++) {
                h_gate_angles_[b] = circuits[b].gates[gate_idx].angle;
            }
            hipMemcpyAsync(d_angles_, h_gate_angles_, batch_size_ * sizeof(double),
                            hipMemcpyHostToDevice, stream_);
            if (gate.type == GateType::RY) {
                apply_ry_gate_batched<<<grid, block, 0, stream_>>>(
                    d_states_, batch_size_, gate.target_qubit, d_angles_, num_qubits_);
            } else if (gate.type == GateType::RZ) {
                apply_rz_gate_batched<<<grid, block, 0, stream_>>>(
                    d_states_, batch_size_, gate.target_qubit, d_angles_, num_qubits_);
            }
        }

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fmt::print(fg(fmt::color::red),
                "Batched gate error at index {}: {}\n",
                gate_idx, hipGetErrorString(err));
            return false;
        }
        ++gate_idx;
    }

    // Don't block here; let measurement synchronize
    return true;
}

bool BatchedQuantumSimulator::measure_all(
    std::vector<std::vector<double>>& expectations)
{
    expectations.resize(batch_size_);
    for (int b = 0; b < batch_size_; b++) {
        expectations[b].resize(num_qubits_, 0.0);
    }
    
    // Zero out device expectations
    hipMemsetAsync(d_expectations_, 0, batch_size_ * num_qubits_ * sizeof(double), stream_);
    
    int block_size = 256;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    
    dim3 grid(num_blocks, num_qubits_, batch_size_);
    dim3 block(block_size);
    
    measure_expectations_batched<<<grid, block, 0, stream_>>>(
        d_states_, batch_size_, d_expectations_, num_qubits_);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Batched measurement error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    // Copy results to pinned host buffer (keeps same stream ordering; kernels finish before copy)
    hipMemcpyAsync(h_expectations_pinned_, d_expectations_,
                    batch_size_ * num_qubits_ * sizeof(double),
                    hipMemcpyDeviceToHost, stream_);
    // Wait for the copy to complete before reshaping
    hipStreamSynchronize(stream_);
    
    // Reshape to 2D
    for (int b = 0; b < batch_size_; b++) {
        for (int q = 0; q < num_qubits_; q++) {
            expectations[b][q] = h_expectations_pinned_[b * num_qubits_ + q];
        }
    }
    
    return true;
}

} // namespace quantum
} // namespace ohmy
