#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Regis Araujo Melo
 * This program is free software under the GPL-3.0 license. See LICENSE file.
 */

#include "ohmy/quantum/batched_cuda_simulator.hpp"
#include <fmt/format.h>

namespace ohmy {
namespace quantum {
namespace cuda {

// Forward declarations of batched kernels
extern __global__ void init_zero_state_batch_kernel(
    Complex* states, int batch_size, size_t state_size);

extern __global__ void apply_rotation_y_batch_kernel(
    Complex* states, int batch_size, int target_qubit, float angle, size_t state_size);

extern __global__ void apply_rotation_z_batch_kernel(
    Complex* states, int batch_size, int target_qubit, float angle, size_t state_size);

extern __global__ void apply_cnot_batch_kernel(
    Complex* states, int batch_size, int control_qubit, int target_qubit, size_t state_size);

extern __global__ void compute_z_expectation_batch_kernel(
    const Complex* states, int batch_size, int target_qubit, size_t state_size, float* expectations);

// --- Constructor ---

BatchedCudaSimulator::BatchedCudaSimulator(int num_qubits, int batch_size, int device_id)
    : num_qubits_(num_qubits)
    , batch_size_(batch_size)
    , device_id_(device_id)
    , state_size_(1ULL << num_qubits)
    , block_size_(DEFAULT_BLOCK_SIZE)
    , d_batch_states_(batch_size * state_size_)
    , d_batch_expectations_(batch_size)
    , compute_stream_()
{
    // Set device
    CUDA_CHECK(hipSetDevice(device_id_));
    
    // Query device
    device_info_ = DeviceInfo::query(device_id_);
    
    if (!device_info_.is_compatible()) {
        throw std::runtime_error(fmt::format(
            "GPU compute capability {}.{} below minimum 7.5",
            device_info_.compute_capability_major,
            device_info_.compute_capability_minor
        ));
    }
    
    // Check memory requirements
    size_t total_memory_needed = batch_size * state_size_ * sizeof(Complex);
    if (device_info_.free_memory < total_memory_needed * 1.2) {
        throw std::runtime_error(fmt::format(
            "Insufficient GPU memory for batch size {}: need {}, have {} free",
            batch_size,
            MemoryRequirements::format_bytes(total_memory_needed * 1.2),
            MemoryRequirements::format_bytes(device_info_.free_memory)
        ));
    }
    
    fmt::print("[CUDA Batch] Initialized {} with {} qubits, batch size {}\n",
               device_info_.name, num_qubits_, batch_size_);
    fmt::print("[CUDA Batch] Memory: {} ({} per state × {} states)\n",
               MemoryRequirements::format_bytes(total_memory_needed),
               MemoryRequirements::format_bytes(state_size_ * sizeof(Complex)),
               batch_size_);
    
    reset_batch();
}

BatchedCudaSimulator::~BatchedCudaSimulator() {
    // RAII handles cleanup
}

// --- Batch Operations ---

void BatchedCudaSimulator::reset_batch() {
    // Initialize all states to |0...0⟩
    dim3 grid = calculate_batch_grid_size(state_size_);
    dim3 block(block_size_);
    
    init_zero_state_batch_kernel<<<grid, block, 0, compute_stream_.get()>>>(
        d_batch_states_.get(), batch_size_, state_size_);
    
    CUDA_CHECK(hipGetLastError());
    compute_stream_.synchronize();
}

void BatchedCudaSimulator::apply_rotation_batch(int qubit, float angle, RotationAxis axis) {
    if (axis == RotationAxis::Y) {
        // R_Y processes pairs of amplitudes
        dim3 grid = calculate_batch_grid_size(state_size_ / 2, true);
        dim3 block(block_size_);
        
        apply_rotation_y_batch_kernel<<<grid, block, 0, compute_stream_.get()>>>(
            d_batch_states_.get(), batch_size_, qubit, angle, state_size_);
    } else {
        // R_Z processes all amplitudes
        dim3 grid = calculate_batch_grid_size(state_size_);
        dim3 block(block_size_);
        
        apply_rotation_z_batch_kernel<<<grid, block, 0, compute_stream_.get()>>>(
            d_batch_states_.get(), batch_size_, qubit, angle, state_size_);
    }
    
    CUDA_CHECK(hipGetLastError());
}

void BatchedCudaSimulator::apply_cnot_batch(int control, int target) {
    dim3 grid = calculate_batch_grid_size(state_size_);
    dim3 block(block_size_);
    
    apply_cnot_batch_kernel<<<grid, block, 0, compute_stream_.get()>>>(
        d_batch_states_.get(), batch_size_, control, target, state_size_);
    
    CUDA_CHECK(hipGetLastError());
}

std::vector<float> BatchedCudaSimulator::measure_batch(const std::vector<int>& qubits) {
    std::vector<float> all_expectations;
    all_expectations.reserve(batch_size_ * qubits.size());
    
    for (int qubit : qubits) {
        // Each state gets its own block for measurement
        dim3 grid(batch_size_);
        dim3 block(block_size_);
        
        compute_z_expectation_batch_kernel<<<grid, block, 0, compute_stream_.get()>>>(
            d_batch_states_.get(), batch_size_, qubit, state_size_, d_batch_expectations_.get());
        
        CUDA_CHECK(hipGetLastError());
        
        // Copy results to host
        std::vector<float> qubit_expectations(batch_size_);
        CUDA_CHECK(hipMemcpy(qubit_expectations.data(), d_batch_expectations_.get(),
                              batch_size_ * sizeof(float), hipMemcpyDeviceToHost));
        
        all_expectations.insert(all_expectations.end(),
                               qubit_expectations.begin(),
                               qubit_expectations.end());
    }
    
    return all_expectations;
}

std::vector<std::vector<Q15>> BatchedCudaSimulator::simulate_and_measure_batch(
    const std::vector<QuantumCircuit>& circuits,
    const std::vector<int>& qubits_to_measure
) {
    if (circuits.empty()) {
        return {};
    }
    
    if (circuits.size() != static_cast<size_t>(batch_size_)) {
        throw std::runtime_error(fmt::format(
            "Circuit count {} != batch size {}",
            circuits.size(), batch_size_
        ));
    }
    
    // Validate all circuits have same structure
    const auto& ref_circuit = circuits[0];
    for (size_t i = 1; i < circuits.size(); i++) {
        if (circuits[i].rotation_gates().size() != ref_circuit.rotation_gates().size() ||
            circuits[i].cnot_gates().size() != ref_circuit.cnot_gates().size()) {
            throw std::runtime_error("All circuits in batch must have same structure");
        }
    }
    
    // Reset all states
    reset_batch();
    
    // Apply gates from reference circuit
    // (In qhash, all circuits have same gates, only nonce differs in hash seed)
    for (const auto& gate : ref_circuit.rotation_gates()) {
        apply_rotation_batch(gate.qubit, static_cast<float>(gate.angle), gate.axis);
    }
    
    for (const auto& gate : ref_circuit.cnot_gates()) {
        apply_cnot_batch(gate.control, gate.target);
    }
    
    // Synchronize before measurement
    compute_stream_.synchronize();
    
    // Measure all qubits for all states
    auto flat_expectations = measure_batch(qubits_to_measure);
    
    // Reshape results: [batch_size][num_qubits]
    std::vector<std::vector<Q15>> results;
    results.reserve(batch_size_);
    
    size_t num_qubits = qubits_to_measure.size();
    for (int batch_idx = 0; batch_idx < batch_size_; batch_idx++) {
        std::vector<Q15> state_expectations;
        state_expectations.reserve(num_qubits);
        
        for (size_t q = 0; q < num_qubits; q++) {
            float expectation = flat_expectations[q * batch_size_ + batch_idx];
            state_expectations.push_back(Q15::from_float(expectation));
        }
        
        results.push_back(std::move(state_expectations));
    }
    
    return results;
}

// --- Helper Methods ---

dim3 BatchedCudaSimulator::calculate_batch_grid_size(
    size_t elements_per_state,
    bool /* is_pair_kernel */  // Unused for now, reserved for future optimization
) const {
    size_t total_elements = batch_size_ * elements_per_state;
    size_t num_blocks = (total_elements + block_size_ - 1) / block_size_;
    return dim3(static_cast<unsigned int>(num_blocks));
}

int BatchedCudaSimulator::get_optimal_batch_size() const {
    size_t memory_per_state = state_size_ * sizeof(Complex);
    size_t usable_memory = device_info_.free_memory * 0.8;  // Leave 20% headroom
    int optimal = static_cast<int>(usable_memory / memory_per_state);
    
    // Clamp to reasonable range
    return std::min(std::max(optimal, 100), MAX_BATCH_SIZE);
}

} // namespace cuda
} // namespace quantum
} // namespace ohmy
