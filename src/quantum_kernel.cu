#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Regis Araujo Melo
 * This program is free software under the GPL-3.0 license. See LICENSE file.
 */

#include "quantum_kernel.cuh"
#include <fmt/core.h>
#include <fmt/color.h>
#include <cmath>

namespace ohmy {
namespace quantum {

// ============================================================================
// CUDA Device Helper Functions
// ============================================================================

/**
 * @brief Complex multiplication
 */
__device__ __forceinline__ Complex complex_mul(Complex a, Complex b) {
    return make_hipDoubleComplex(
        hipCreal(a) * hipCreal(b) - hipCimag(a) * hipCimag(b),
        hipCreal(a) * hipCimag(b) + hipCimag(a) * hipCreal(b)
    );
}

/**
 * @brief Complex addition
 */
__device__ __forceinline__ Complex complex_add(Complex a, Complex b) {
    return make_hipDoubleComplex(
        hipCreal(a) + hipCreal(b),
        hipCimag(a) + hipCimag(b)
    );
}

/**
 * @brief Check if qubit is set in basis state index
 */
__device__ __forceinline__ bool is_qubit_set(size_t index, int qubit) {
    return (index & (1ULL << qubit)) != 0;
}

/**
 * @brief Flip qubit bit in basis state index
 */
__device__ __forceinline__ size_t flip_qubit(size_t index, int qubit) {
    return index ^ (1ULL << qubit);
}

// ============================================================================
// CUDA Kernels
// ============================================================================

__global__ void init_quantum_state(Complex* state, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        // Initialize to |0⟩^⊗n: first element = 1, rest = 0
        if (idx == 0) {
            state[idx] = make_hipDoubleComplex(1.0, 0.0);
        } else {
            state[idx] = make_hipDoubleComplex(0.0, 0.0);
        }
    }
}

__global__ void apply_rx_gate(Complex* state, int target_qubit, double angle, int num_qubits) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t state_size = 1ULL << num_qubits;
    
    if (idx >= state_size) return;
    
    // Only process pairs where target qubit is |0⟩
    if (is_qubit_set(idx, target_qubit)) return;
    
    size_t idx_flipped = flip_qubit(idx, target_qubit);
    
    // Rx(θ) = cos(θ/2)*I - i*sin(θ/2)*X
    double cos_half = cos(angle / 2.0);
    double sin_half = sin(angle / 2.0);
    
    Complex state0 = state[idx];
    Complex state1 = state[idx_flipped];
    
    // New |0⟩ component: cos(θ/2)|0⟩ - i*sin(θ/2)|1⟩
    state[idx] = make_hipDoubleComplex(
        cos_half * hipCreal(state0) + sin_half * hipCimag(state1),
        cos_half * hipCimag(state0) - sin_half * hipCreal(state1)
    );
    
    // New |1⟩ component: -i*sin(θ/2)|0⟩ + cos(θ/2)|1⟩
    state[idx_flipped] = make_hipDoubleComplex(
        cos_half * hipCreal(state1) + sin_half * hipCimag(state0),
        cos_half * hipCimag(state1) - sin_half * hipCreal(state0)
    );
}

__global__ void apply_ry_gate(Complex* state, int target_qubit, double angle, int num_qubits) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t state_size = 1ULL << num_qubits;
    
    if (idx >= state_size) return;
    
    // Only process pairs where target qubit is |0⟩
    if (is_qubit_set(idx, target_qubit)) return;
    
    size_t idx_flipped = flip_qubit(idx, target_qubit);
    
    // Ry(θ) = cos(θ/2)*I - i*sin(θ/2)*Y
    double cos_half = cos(angle / 2.0);
    double sin_half = sin(angle / 2.0);
    
    Complex state0 = state[idx];
    Complex state1 = state[idx_flipped];
    
    // New |0⟩ component: cos(θ/2)|0⟩ - sin(θ/2)|1⟩
    state[idx] = make_hipDoubleComplex(
        cos_half * hipCreal(state0) - sin_half * hipCreal(state1),
        cos_half * hipCimag(state0) - sin_half * hipCimag(state1)
    );
    
    // New |1⟩ component: sin(θ/2)|0⟩ + cos(θ/2)|1⟩
    state[idx_flipped] = make_hipDoubleComplex(
        sin_half * hipCreal(state0) + cos_half * hipCreal(state1),
        sin_half * hipCimag(state0) + cos_half * hipCimag(state1)
    );
}

__global__ void apply_rz_gate(Complex* state, int target_qubit, double angle, int num_qubits) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t state_size = 1ULL << num_qubits;
    
    if (idx >= state_size) return;
    
    // Rz(θ) = e^(-iθ/2)|0⟩⟨0| + e^(iθ/2)|1⟩⟨1|
    double half_angle = angle / 2.0;
    
    Complex phase;
    if (is_qubit_set(idx, target_qubit)) {
        // |1⟩: multiply by e^(iθ/2) = cos(θ/2) + i*sin(θ/2)
        phase = make_hipDoubleComplex(cos(half_angle), sin(half_angle));
    } else {
        // |0⟩: multiply by e^(-iθ/2) = cos(θ/2) - i*sin(θ/2)
        phase = make_hipDoubleComplex(cos(half_angle), -sin(half_angle));
    }
    
    state[idx] = complex_mul(state[idx], phase);
}

__global__ void apply_cnot_gate(Complex* state, int control_qubit, int target_qubit, int num_qubits) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t state_size = 1ULL << num_qubits;
    
    if (idx >= state_size) return;
    
    // CNOT flips target only when control is |1⟩
    if (!is_qubit_set(idx, control_qubit)) return;
    if (is_qubit_set(idx, target_qubit)) return;
    
    size_t idx_flipped = flip_qubit(idx, target_qubit);
    
    // Swap amplitudes
    Complex temp = state[idx];
    state[idx] = state[idx_flipped];
    state[idx_flipped] = temp;
}

/**
 * @brief OPTIMIZED: Fused RY+RZ layer kernel
 * 
 * This kernel applies RY(θy) followed by RZ(θz) to all qubits in a single pass.
 * Instead of 32 separate kernel launches (16 RY + 16 RZ), we do everything in
 * one kernel, dramatically reducing launch overhead.
 * 
 * Mathematical operation per qubit q:
 *   |ψ'⟩ = Rz(θz[q]) * Ry(θy[q]) * |ψ⟩
 * 
 * Combined rotation matrix:
 *   Rz*Ry = [ cos(θy/2)*e^(-iθz/2)    -sin(θy/2)*e^(-iθz/2) ]
 *           [ sin(θy/2)*e^(iθz/2)      cos(θy/2)*e^(iθz/2)  ]
 * 
 * Strategy:
 * - Each thread processes one basis state |i⟩
 * - For each qubit q, compute local 2x2 rotation for that qubit's subspace
 * - Apply rotations sequentially to maintain determinism
 */
/**
 * @brief OPTIMIZED: Fused RY+RZ kernel for a SINGLE qubit
 * 
 * This kernel applies RY(θy) followed by RZ(θz) to ONE qubit in a single pass.
 * Mathematical operation: |ψ'⟩ = Rz(θz) * Ry(θy) * |ψ⟩
 * 
 * We still need to call this 16 times (once per qubit), but each call fuses
 * 2 operations that would normally require 2 separate kernels.
 * 
 * Net result: 32 kernel launches (16 RY + 16 RZ) → 16 kernel launches (2x reduction)
 * 
 * This is the CORRECT approach that maintains determinism and avoids race conditions.
 */
__global__ void apply_ry_rz_fused_single_qubit(
    Complex* state,
    double theta_y,
    double theta_z,
    int qubit,
    int num_qubits
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t state_size = 1ULL << num_qubits;
    
    if (idx >= state_size) return;
    
    // Only process pairs where this basis state has qubit = |0⟩
    if (is_qubit_set(idx, qubit)) return;
    
    size_t idx_flipped = flip_qubit(idx, qubit);
    
    // Precompute rotation matrix elements
    double cos_y_half = cos(theta_y / 2.0);
    double sin_y_half = sin(theta_y / 2.0);
    double cos_z_half = cos(theta_z / 2.0);
    double sin_z_half = sin(theta_z / 2.0);
    
    // Get current amplitudes
    Complex amp0 = state[idx];
    Complex amp1 = state[idx_flipped];
    
    // Apply fused Rz*Ry rotation
    // Matrix elements: [[ cos(y/2)*e^(-iz/2), -sin(y/2)*e^(-iz/2) ],
    //                   [ sin(y/2)*e^(iz/2),   cos(y/2)*e^(iz/2)  ]]
    
    Complex e_minus = make_hipDoubleComplex(cos_z_half, -sin_z_half);
    Complex e_plus = make_hipDoubleComplex(cos_z_half, sin_z_half);
    
    // |0⟩ component
    Complex term0_0 = complex_mul(make_hipDoubleComplex(cos_y_half, 0), e_minus);
    Complex term0_1 = complex_mul(make_hipDoubleComplex(-sin_y_half, 0), e_minus);
    Complex new_amp0 = complex_add(
        complex_mul(term0_0, amp0),
        complex_mul(term0_1, amp1)
    );
    
    // |1⟩ component
    Complex term1_0 = complex_mul(make_hipDoubleComplex(sin_y_half, 0), e_plus);
    Complex term1_1 = complex_mul(make_hipDoubleComplex(cos_y_half, 0), e_plus);
    Complex new_amp1 = complex_add(
        complex_mul(term1_0, amp0),
        complex_mul(term1_1, amp1)
    );
    
    // Write back (no race condition - each thread owns its idx pair)
    state[idx] = new_amp0;
    state[idx_flipped] = new_amp1;
}

__global__ void measure_expectations(const Complex* state, double* expectations, int num_qubits) {
    int qubit = blockIdx.x;
    if (qubit >= num_qubits) return;
    
    size_t state_size = 1ULL << num_qubits;
    int tid = threadIdx.x;
    int block_size = blockDim.x;
    
    // Shared memory for reduction
    __shared__ double shared_expectation[256];
    
    double local_sum = 0.0;
    
    // Each thread processes multiple basis states
    for (size_t idx = tid; idx < state_size; idx += block_size) {
        double prob = hipCreal(state[idx]) * hipCreal(state[idx]) + 
                     hipCimag(state[idx]) * hipCimag(state[idx]);
        
        // ⟨σz⟩ = P(|0⟩) - P(|1⟩)
        if (is_qubit_set(idx, qubit)) {
            local_sum -= prob;  // |1⟩ contributes -1
        } else {
            local_sum += prob;  // |0⟩ contributes +1
        }
    }
    
    shared_expectation[tid] = local_sum;
    __syncthreads();
    
    // Reduction in shared memory
    for (int stride = block_size / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_expectation[tid] += shared_expectation[tid + stride];
        }
        __syncthreads();
    }
    
    // Thread 0 writes result
    if (tid == 0) {
        expectations[qubit] = shared_expectation[0];
    }
}

// ============================================================================
// Host-side QuantumSimulator Implementation
// ============================================================================

QuantumSimulator::QuantumSimulator(int num_qubits)
    : num_qubits_(num_qubits)
    , state_size_(1ULL << num_qubits)
    , d_state_(nullptr)
    , d_expectations_(nullptr) {
    
    // Allocate device memory for state vector
    size_t state_bytes = state_size_ * sizeof(Complex);
    hipError_t err = hipMalloc(&d_state_, state_bytes);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red) | fmt::emphasis::bold,
            "Failed to allocate GPU memory for quantum state: {}\n",
            hipGetErrorString(err));
        throw std::runtime_error("CUDA allocation failed");
    }
    
    // Allocate device memory for expectations
    err = hipMalloc(&d_expectations_, num_qubits_ * sizeof(double));
    if (err != hipSuccess) {
        hipFree(d_state_);
        fmt::print(fg(fmt::color::red) | fmt::emphasis::bold,
            "Failed to allocate GPU memory for expectations: {}\n",
            hipGetErrorString(err));
        throw std::runtime_error("CUDA allocation failed");
    }
    
    fmt::print(fg(fmt::color::green),
        "Quantum simulator initialized: {} qubits, state size = {}\n",
        num_qubits_, state_size_);
}

QuantumSimulator::~QuantumSimulator() {
    if (d_state_) {
        hipFree(d_state_);
    }
    if (d_expectations_) {
        hipFree(d_expectations_);
    }
}

bool QuantumSimulator::initialize_state() {
    // Launch kernel to initialize state to |0⟩^⊗n
    int block_size = 256;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    
    init_quantum_state<<<num_blocks, block_size>>>(d_state_, state_size_);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Kernel launch error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Kernel execution error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    return true;
}

bool QuantumSimulator::apply_circuit(const QuantumCircuit& circuit) {
    if (circuit.num_qubits != num_qubits_) {
        fmt::print(fg(fmt::color::red),
            "Circuit qubit count mismatch: {} vs {}\n",
            circuit.num_qubits, num_qubits_);
        return false;
    }
    
    int block_size = 256;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    
    // Apply gates sequentially
    for (const auto& gate : circuit.gates) {
        switch (gate.type) {
            case GateType::RX:
                apply_rx_gate<<<num_blocks, block_size>>>(
                    d_state_, gate.target_qubit, gate.angle, num_qubits_);
                break;
                
            case GateType::RY:
                apply_ry_gate<<<num_blocks, block_size>>>(
                    d_state_, gate.target_qubit, gate.angle, num_qubits_);
                break;
                
            case GateType::RZ:
                apply_rz_gate<<<num_blocks, block_size>>>(
                    d_state_, gate.target_qubit, gate.angle, num_qubits_);
                break;
                
            case GateType::CNOT:
                apply_cnot_gate<<<num_blocks, block_size>>>(
                    d_state_, gate.control_qubit, gate.target_qubit, num_qubits_);
                break;
        }
        
        // Check for errors after each gate
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fmt::print(fg(fmt::color::red),
                "Gate kernel launch error: {}\n", hipGetErrorString(err));
            return false;
        }
    }
    
    // Synchronize after all gates
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Circuit execution error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    return true;
}

bool QuantumSimulator::apply_circuit_optimized(const QuantumCircuit& circuit) {
    if (circuit.num_qubits != num_qubits_) {
        fmt::print(fg(fmt::color::red),
            "Circuit qubit count mismatch: {} vs {}\n",
            circuit.num_qubits, num_qubits_);
        return false;
    }
    
    hipError_t err;
    int block_size = 256;
    int num_blocks = (state_size_ + block_size - 1) / block_size;
    
    // Process circuit gates with optimizations
    // QTC circuit structure: [RY_all → RZ_all → CNOT_chain] × 2 layers
    // OPTIMIZATIONS:
    // 1. Fuse adjacent RY+RZ pairs per qubit (32 launches → 16 launches)
    // 2. Detect and optimize CNOT chains with shared memory (15 launches → 1 launch per chain)
    
    size_t gate_idx = 0;
    const size_t total_gates = circuit.gates.size();
    
    while (gate_idx < total_gates) {
        // OPTIMIZATION 1: Try to detect RY-RZ pair on the same qubit
        if (gate_idx + 1 < total_gates &&
            circuit.gates[gate_idx].type == GateType::RY &&
            circuit.gates[gate_idx + 1].type == GateType::RZ &&
            circuit.gates[gate_idx].target_qubit == circuit.gates[gate_idx + 1].target_qubit) {
            
            // Found fusible pair!
            int qubit = circuit.gates[gate_idx].target_qubit;
            double theta_y = circuit.gates[gate_idx].angle;
            double theta_z = circuit.gates[gate_idx + 1].angle;
            
            // Launch fused kernel
            apply_ry_rz_fused_single_qubit<<<num_blocks, block_size>>>(
                d_state_, theta_y, theta_z, qubit, num_qubits_);
            
            err = hipGetLastError();
            if (err != hipSuccess) {
                fmt::print(fg(fmt::color::red),
                    "Fused RY+RZ kernel error: {}\n", hipGetErrorString(err));
                return false;
            }
            
            gate_idx += 2;  // Skip both gates
        }
        // Apply CNOT gates (no chain optimization - complexity not worth it)
        else if (circuit.gates[gate_idx].type == GateType::CNOT) {
            const auto& gate = circuit.gates[gate_idx];
            apply_cnot_gate<<<num_blocks, block_size>>>(
                d_state_, gate.control_qubit, gate.target_qubit, num_qubits_);
            
            err = hipGetLastError();
            if (err != hipSuccess) {
                fmt::print(fg(fmt::color::red),
                    "CNOT kernel error: {}\n", hipGetErrorString(err));
                return false;
            }
            
            gate_idx++;
        }
        else {
            // Non-fusible gate - apply individually
            const auto& gate = circuit.gates[gate_idx];
            
            switch (gate.type) {
                case GateType::RX:
                    apply_rx_gate<<<num_blocks, block_size>>>(
                        d_state_, gate.target_qubit, gate.angle, num_qubits_);
                    break;
                case GateType::RY:
                    apply_ry_gate<<<num_blocks, block_size>>>(
                        d_state_, gate.target_qubit, gate.angle, num_qubits_);
                    break;
                case GateType::RZ:
                    apply_rz_gate<<<num_blocks, block_size>>>(
                        d_state_, gate.target_qubit, gate.angle, num_qubits_);
                    break;
                default:
                    break;
            }
            
            err = hipGetLastError();
            if (err != hipSuccess) {
                fmt::print(fg(fmt::color::red),
                    "Gate kernel error: {}\n", hipGetErrorString(err));
                return false;
            }
            
            gate_idx++;
        }
    }
    
    // Synchronize
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Circuit execution error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    return true;
}

bool QuantumSimulator::measure(std::vector<double>& expectations) {
    expectations.resize(num_qubits_);
    
    int block_size = 256;
    
    // Launch one block per qubit
    measure_expectations<<<num_qubits_, block_size>>>(
        d_state_, d_expectations_, num_qubits_);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Measure kernel launch error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Measure execution error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    // Copy results back to host
    err = hipMemcpy(expectations.data(), d_expectations_,
                     num_qubits_ * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fmt::print(fg(fmt::color::red),
            "Measure memcpy error: {}\n", hipGetErrorString(err));
        return false;
    }
    
    return true;
}

} // namespace quantum
} // namespace ohmy
